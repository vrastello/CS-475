#include "hip/hip_runtime.h"
#include "exception.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "helper_image.h"
#include "helper_string.h"
#include "helper_timer.h"

#ifndef NUMTRIALS
#define NUMTRIALS	(1024 * 1024)
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE	512
#endif

#ifndef NUMBLOCKS
#define NUMBLOCKS    (NUMTRIALS/BLOCKSIZE)
#endif

// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN =     70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL =       5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL

// function prototypes:
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );
void            CudaCheckError( );

// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
        return (M_PI/180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
        unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

        // randomize everything:
        float v   = dvs[gid];
        float thr = Radians( dths[gid] );
        float vx  = v * cos(thr);
        float vy  = v * sin(thr);
        float  g  =  dgs[gid];
        float  h  =  dhs[gid];
        float  d  =  dds[gid];

        int numHits = 0;

        // see if the ball doesn't even reach the cliff:
        float t = -vy / ( 0.5*GRAVITY );
        float x = vx * t;
        if( x > g) 
        {
                // see if the ball hits the vertical cliff face:
                t = g/vx;
                float t_squared = pow(t, 2);
                float y = vy*t + 0.5 * GRAVITY * t_squared;
                if( y > h )
                {
                        // the ball hits the upper deck:
                        // the time solution for this is a quadratic equation of the form:
                        // at^2 + bt + c = 0.
                        // where 'a' multiplies time^2
                        //       'b' multiplies time
                        //       'c' is a constant
                        float a = 0.5 * GRAVITY;
                        float b = vy;
                        float c = -h;
                        float disc = b*b - 4.f*a*c;	// quadratic formula discriminant

                        // successfully hits the ground above the cliff:
                        // get the intersection:
                        disc = sqrtf( disc );
                        float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
                        float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

                        // only care about the second intersection
                        float tmax = t1;
                        if( t2 > t1 )
                                tmax = t2;

                        // how far does the ball land horizontlly from the edge of the cliff?
                        float upperDist = vx * tmax  -  g;

                        // see if the ball hits the castle:
                        if(  fabs( upperDist - d ) <= TOL )
                        {
                                numHits = 1;
                        }
                } // if ball clears the cliff face
	} // if ball gets as far as the cliff face
                dhits[gid] = numHits;
} // for( # of  monte carlo trials )
                                
        

// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

int
main( int argc, char* argv[ ] )
{
        TimeOfDaySeed( );

        //int dev = findCudaDevice(argc, (const char **)argv);

        // better to define these here so that the rand() calls don't get into the thread timing:
        float *hvs   = new float [NUMTRIALS];
        float *hths  = new float [NUMTRIALS];
        float *hgs   = new float [NUMTRIALS];
        float *hhs   = new float [NUMTRIALS];
        float *hds   = new float [NUMTRIALS];
        int   *hhits = new int   [NUMTRIALS];

        // fill the random-value arrays:

        for( int n = 0; n < NUMTRIALS; n++ )
	{
		hvs[n]  = Ranf(  VMIN,  VMAX );
		hths[n] = Ranf( THMIN, THMAX );
 		hgs[n]  = Ranf(  GMIN,  GMAX );
 		hhs[n]  = Ranf(  HMIN,  HMAX );
 		hds[n]  = Ranf(  DMIN,  DMAX );
	}


        // allocate device memory:
        float *dvs, *dths, *dgs, *dhs, *dds;
        int   *dhits;

        hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
        hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
        CudaCheckError( );

        // copy host memory to the device:
        hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        CudaCheckError( );

        // setup the execution parameters:
        dim3 grid( NUMBLOCKS, 1, 1 );
        dim3 threads( BLOCKSIZE, 1, 1 );

        // allocate cuda events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop  );
        CudaCheckError( );

        // let the gpu go quiet:
        hipDeviceSynchronize( );

        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

        // execute the kernel:
        MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds,   OUT dhits );

        // record the stop event:
        hipEventRecord( stop, NULL );
        CudaCheckError( );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
        hipEventSynchronize( stop );
        CudaCheckError( );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, start, stop );
        CudaCheckError( );

	// compute and print the performance
        double secondsTotal = 0.001 * (double)msecTotal;
        double multsPerSecond = (double)NUMTRIALS / secondsTotal;
        double megaMultsPerSecond = multsPerSecond / 1000000.;

        // copy result from the device to the host:
        hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
        CudaCheckError( );

        // add up the hhits[ ] array: :

        int total_hits = 0;

	for( int n = 0; n < NUMTRIALS; n++ )
	{
                total_hits += hhits[n];
	}

        // compute and print the probability:

        float probability = (float)total_hits/(float)(NUMTRIALS);
        fprintf( stderr, "%12d\t%4d\t%10.2lf\t%6.2f%%\n", NUMTRIALS, 
        BLOCKSIZE, megaMultsPerSecond, 100.*probability);

        // clean up host memory:
        delete [ ] hvs;
        delete [ ] hths;
        delete [ ] hgs;
        delete [ ] hhs;
        delete [ ] hds;
        delete [ ] hhits;

        // clean up device memory:
        hipFree( dvs );
        hipFree( dths );
        hipFree( dgs );
        hipFree( dhs );
        hipFree( dds );
        hipFree( dhits );
        CudaCheckError( );

	return 0;
}

void
CudaCheckError( )
{
        hipError_t e = hipGetLastError( );
        if( e != hipSuccess )
        {
                fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
        }
}

float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}
